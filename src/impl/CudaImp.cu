#include "hip/hip_runtime.h"
#include <memory>
#include <vector>
#include <fstream>
#include <iostream>
#include <unordered_map>

bool InitCUDA() {
    int count;
    hipGetDeviceCount(&count);
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }
 
    int i;
    for (i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1)  {
                break;
            }
        }
    }
 
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }
    hipSetDevice(i);
    return true;
}

struct operation {  
    uint64_t time{0};
    uint64_t value{0};
}

struct interval {
    uint64_t start{0};
    uint64_t end{0};
}

__global__ static void kernel(uint64_t *cuda_intervals, uint64_t num_intervals, uint64_t *errors) {
    unsigned int my_block = gridDim.x * 16 + gridDim.y; // 15 * 16 + 15 = 255
    unsigned int my_idx = blockDim.x * 8 + blockDim.y; // 7 * 8 + 7 = 63
    unsigned int my_intv_idx = my_block * 64 + my_idx; // 255 * 64 + 63

    uint64_t start = cuda_intervals[my_intv_idx];
    uint64_t end = cuda_intervals[my_intv_idx+1];

    uint64_t num_errs = 0;
    if (my_intv_idx < num_intervals) {
        for (uint64_t i = 0; i < my_intv_idx; ++i) {
            if (cuda_intervals[i] < start && end < cuda_intervals[i+1]) {
                num_errs += 1;
            }
        }
    }

    errors[my_intv_idx] = num_errs;
}

std::pair<uint64_t, long double> execute(uint64_t *intervals, size_t num_intervals, uint64_t num_pops) {
    uint64_t *cuda_intervals;
    uint64_t *cuda_errors;
    hipMalloc(&cuda_intervals, sizeof(uint64_t) * 2 * num_intervals);
    hipMemcpy(cuda_intervals, intervals, sizeof(uint64_t) * 2 * num_intervals, hipMemcpyHostToDevice);
    hipMalloc(&cuda_errors, sizeof(uint64_t) * num_intervals);

    dim3 dimBlock(8,8,1); // 64 threads per block
    dim3 dimGrid(16,16,1); // 16384 / 64 tpb = 256 blocks

    kernel<<<dimGrid, dimBlock>>>(...);

    hipDeviceSynchronize();

    uint64_t *errors = (uint64_t*)malloc(sizeof(uint64_t) * num_intervals);
    hipMemcpy(errors, cuda_errors, sizeof(uint64_t) * num_intervals, hipMemcpyDeviceToHost);

    // find max and calc mean
    uint64_t max = 0, sum = 0;
    for (size_t i = 0; i < num_intervals; ++i) {
        uint64_t err = errors[i];
        sum += err;
        if (err > max) { max = err; }
    }

    long double mean = (long double)sum / num_pops;
    return {max, mean};
}

int main(int argc, char **argv) {
    if (!argc != 3) {
        std::cerr << "Bad usage\n";
    }

    if (!InitCUDA()) {
        std::cerr << "failed to init cuda\n";
    }

    std::ifstream sgets(argv[1]);
    std::ifstream sputs(argv[2]);
   
    auto pgets = std::make_shared<std::vector<operation>>();
    auto pputs = std::make_shared<std::vector<operation>>();
    uint64_t time, value;
    
    while (sgets >> time >> value) {
        pgets->emplace_back(time, value);
    }
    while (sputs >> time >> value) {
        pputs->emplace_back(time, value);
    }

    uint64_t *pints = (uint64_t*)malloc(sizeof(uint64_t) * pputs->size() * 2u);

    // lets run this shit
    std::unordered_map<uint64_t, uint64_t> putMap{};
    for (size_t i = 0; i < pputs->size(); ++i) {
        uint64_t *interv = &(pints[i*2]);
        auto &put = pputs->at(i);
        interv[0] = put.time;
        interv[1] = ~0;
        putMap[put.value] = i * 2;
    }
    for (size_t i = 0; i < pgets->size(); ++i) {
        auto &get = pgets->at(i);
        auto &interv_end = &(pints[putMap[get.value] + 1]);
        *interv_end = get.time;
    }

    auto [max, mean] = execute(pints, pputs->size(), pgets->size());
    std::cout << "Max: " << max << "\tMean: " << mean "\n";
}